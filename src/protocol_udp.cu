#include "hip/hip_runtime.h"


#include <doca_gpunetio_dev_buf.cuh>
#include <doca_gpunetio_dev_eth_rxq.cuh>
#include <doca_gpunetio_dev_eth_txq.cuh>
#include <doca_gpunetio_dev_sem.cuh>

#include "lng/doca-util.h"

#include <vector>

DOCA_LOG_REGISTER(PROT_UDP);

namespace lng {

__device__ __inline__ int
raw_to_udp(const uintptr_t buf_addr, struct eth_ip_udp_hdr** hdr, uint8_t** payload)
{
    (*hdr) = (struct eth_ip_udp_hdr*)buf_addr;
    (*payload) = (uint8_t*)(buf_addr + sizeof(struct eth_ip_udp_hdr));

    return 0;
}

__global__ void cuda_kernel_receive_udp(
    struct doca_gpu_eth_rxq* rxq,
    int sem_num,
    struct doca_gpu_semaphore_gpu* sem_recvinfo,
    bool is_warmup)
{
    if (is_warmup) {
        if (threadIdx.x == 0) {
            printf("warmup cuda_kernel_receive_udp\n");
        }
        return;
    }

    __shared__ uint32_t rx_pkt_num;
    __shared__ uint64_t rx_buf_idx;

    // __shared__ bool is_fin;
    uint32_t clock_count = 0;

    doca_error_t ret;
    struct rx_info* rx_info_global;
    struct doca_gpu_buf* buf_ptr;
    struct eth_ip_udp_hdr* hdr;
    uintptr_t buf_addr;
    uint64_t buf_idx = 0;
    uint32_t laneId = threadIdx.x % WARP_SIZE;
    uint32_t warp_id = threadIdx.x / WARP_SIZE;
    uint32_t sem_stats_idx = 0;
    uint8_t* payload;
    uint32_t max_pkts;
    uint64_t timeout_ns;
    uint64_t doca_gpu_buf_idx = laneId;

    __shared__ enum doca_gpu_semaphore_status rx_status;

    max_pkts = MAX_RX_NUM_PKTS;
    timeout_ns = MAX_RX_TIMEOUT_NS;

    if (blockIdx.x >= 2) {
        return;
    }

    __syncthreads();

    if (blockIdx.x == 0) {

        while (true) {

            ret = doca_gpu_dev_eth_rxq_receive_block(rxq, max_pkts, timeout_ns, &rx_pkt_num, &rx_buf_idx);
            /* If any thread returns receive error, the whole execution stops */
            if (ret != DOCA_SUCCESS) {
                if (threadIdx.x == 0) {
                    /*
                     * printf in CUDA kernel may be a good idea only to report critical errors or debugging.
                     * If application prints this message on the console, something bad happened and
                     * applications needs to exit
                     */
                    printf("Receive TCP kernel error %d Block %d rxpkts %d error %d\n", ret, blockIdx.x, rx_pkt_num, ret);
                    // DOCA_GPUNETIO_VOLATILE(*exit_cond) = 1;
                }
                break;
            }
            // }

            if (rx_pkt_num == 0)
                continue;

            __syncthreads();

            if (threadIdx.x == 0 && rx_pkt_num > 0) {

                while (true) {
                    doca_gpu_dev_semaphore_get_status(sem_recvinfo, sem_stats_idx, &rx_status);

                    if (rx_status == DOCA_GPU_SEMAPHORE_STATUS_FREE) {
                        ret = doca_gpu_dev_semaphore_get_custom_info_addr(sem_recvinfo, sem_stats_idx, (void**)&rx_info_global);
                        if (ret != DOCA_SUCCESS) {
                            printf("TCP Error %d doca_gpu_dev_semaphore_get_custom_info_addr block %d thread %d\n", ret, blockIdx.x, threadIdx.x);
                            // DOCA_GPUNETIO_VOLATILE(*exit_cond) = 1;
                            break;
                        }
                        DOCA_GPUNETIO_VOLATILE(rx_info_global->rx_pkt_num) = DOCA_GPUNETIO_VOLATILE(rx_pkt_num);
                        DOCA_GPUNETIO_VOLATILE(rx_info_global->rx_buf_idx) = DOCA_GPUNETIO_VOLATILE(rx_buf_idx);
                        // printf("%d rx_pkt_num recv\n", rx_pkt_num);
                        // printf("%d rx_buf_idx recv\n", rx_buf_idx);

                        // __threadfence();
                        doca_gpu_dev_semaphore_set_status(sem_recvinfo, sem_stats_idx, DOCA_GPU_SEMAPHORE_STATUS_READY);
                        if (ret != DOCA_SUCCESS) {
                            printf("TCP semaphore recv error\n");
                            return;
                        }
                        __threadfence_system();
                        break;
                    } else {
                        printf("not good to reach here\n");
                    }
                }

                sem_stats_idx = (sem_stats_idx + 1) % sem_num;
            }
            __syncthreads();
        }
    }
}

__global__ void cuda_kernel_makeframe_echo_server(
    struct doca_gpu_eth_rxq* rxq,
    int sem_rx_num, struct doca_gpu_semaphore_gpu* sem_rx_recvinfo,
    int sem_fr_num, struct doca_gpu_semaphore_gpu* sem_fr_recvinfo,
    bool is_warmup)
{
    if (is_warmup) {
        if (threadIdx.x == 0) {
            printf("warmup cuda_kernel_makeframe\n");
        }
        return;
    }
    // printf("cuda_kernel_makeframe run\n");
    __shared__ uint32_t rx_pkt_num;
    __shared__ uint64_t rx_buf_idx;

    __shared__ bool packet_reached;

    __shared__ uint64_t frame_head;

    // __shared__ uint64_t tar_buf_total_size;

    doca_error_t ret;
    struct doca_gpu_buf* buf_ptr;
    struct rx_info* rx_info_global;
    struct fr_info* fr_info_global;
    struct eth_ip_udp_hdr* hdr;
    uintptr_t buf_addr;
    uint32_t sem_rx_recvinfo_idx = 0;
    __shared__ uint32_t sem_fr_recvinfo_idx;
    uint8_t* payload;
    __shared__ bool all_frame_done;

    if (threadIdx.x == 0) {
        frame_head = 0;
        packet_reached = false;
        all_frame_done = false;
        sem_fr_recvinfo_idx = 0;
    }

    if (blockIdx.x != 0) {
        return;
    }

    __syncthreads();

    __shared__ enum doca_gpu_semaphore_status rx_status;
    __shared__ enum doca_gpu_semaphore_status fr_status;

    while (true) {

        if (threadIdx.x == 0) {
            while (!packet_reached) {
                ret = doca_gpu_dev_semaphore_get_status(sem_rx_recvinfo, sem_rx_recvinfo_idx, &rx_status);
                if (ret != DOCA_SUCCESS) {
                    printf("TCP semaphore error");
                    return;
                }
                if (rx_status == DOCA_GPU_SEMAPHORE_STATUS_READY) {

                    ret = doca_gpu_dev_semaphore_get_custom_info_addr(sem_rx_recvinfo, sem_rx_recvinfo_idx, (void**)&(rx_info_global));
                    if (ret != DOCA_SUCCESS) {
                        printf("TCP semaphore get address error\n");
                        return;
                    }

                    DOCA_GPUNETIO_VOLATILE(rx_pkt_num) = DOCA_GPUNETIO_VOLATILE(rx_info_global->rx_pkt_num);
                    DOCA_GPUNETIO_VOLATILE(rx_buf_idx) = DOCA_GPUNETIO_VOLATILE(rx_info_global->rx_buf_idx);

                    __threadfence();

                    // printf("%d rx_pkt_num frame \n", rx_pkt_num);
                    // printf("%d rx_buf_idx frame \n", rx_buf_idx);

                    ret = doca_gpu_dev_semaphore_set_status(sem_rx_recvinfo, sem_rx_recvinfo_idx, DOCA_GPU_SEMAPHORE_STATUS_FREE);
                    if (ret != DOCA_SUCCESS) {
                        printf("TCP semaphore error\n");
                        return;
                    }
                    __threadfence_system();
                    packet_reached = true;

                    sem_rx_recvinfo_idx = (sem_rx_recvinfo_idx + 1) % sem_rx_num;
                } else {
                    rx_pkt_num = 0;
                }
            }
        }

        __syncthreads();

        if (!packet_reached)
            continue;

        __syncthreads();

        uint32_t tail_sem_idx = (sem_fr_recvinfo_idx + rx_pkt_num - 1) % sem_fr_num;

        while (!all_frame_done) {
            ret = doca_gpu_dev_semaphore_get_status(sem_fr_recvinfo, tail_sem_idx, &fr_status);
            if (ret != DOCA_SUCCESS) {
                printf("fr semaphore failed.");
                return;
            }
            if (fr_status == DOCA_GPU_SEMAPHORE_STATUS_FREE) {
                for (uint64_t idx = rx_buf_idx + threadIdx.x,
                              sem_idx = sem_fr_recvinfo_idx + threadIdx.x;
                     idx < rx_buf_idx + rx_pkt_num;
                     idx += blockDim.x, sem_idx += blockDim.x) {
                    ret = doca_gpu_dev_eth_rxq_get_buf(rxq, idx, &buf_ptr);
                    if (ret != DOCA_SUCCESS) {
                        printf("TCP Error %d doca_gpu_dev_eth_rxq_get_buf block %d thread %d\n", ret, blockIdx.x, threadIdx.x);
                        // DOCA_GPUNETIO_VOLATILE(*exit_cond) = 1;
                        break;
                    }
                    ret = doca_gpu_dev_buf_get_addr(buf_ptr, &buf_addr);
                    if (ret != DOCA_SUCCESS) {
                        printf("TCP Error %d doca_gpu_dev_eth_rxq_get_buf block %d thread %d\n", ret, blockIdx.x, threadIdx.x);
                        // DOCA_GPUNETIO_VOLATILE(*exit_cond) = 1;
                        break;
                    }

                    ret = doca_gpu_dev_semaphore_get_custom_info_addr(sem_fr_recvinfo, sem_idx % sem_fr_num, (void**)&(fr_info_global));
                    if (ret != DOCA_SUCCESS) {
                        printf("TCP semaphore get address error\n");
                        return;
                    }
                    uint8_t* payload = (uint8_t*)buf_addr;

                    DOCA_GPUNETIO_VOLATILE(fr_info_global->eth_payload) = DOCA_GPUNETIO_VOLATILE(payload);

                    __threadfence();

                    // printf("%d rx_pkt_num frame \n", rx_pkt_num);
                    // printf("%d rx_buf_idx frame \n", rx_buf_idx);

                    ret = doca_gpu_dev_semaphore_set_status(sem_fr_recvinfo, sem_idx % sem_fr_num, DOCA_GPU_SEMAPHORE_STATUS_READY);

                    raw_to_udp(buf_addr, &hdr, &payload);
                    // printf("%d l4_hdr bytes recv\n", BYTE_SWAP16(hdr->l4_hdr.dgram_len));
                    // printf("%d l3_hdr\n", BYTE_SWAP16(hdr->l3_hdr.total_length));
                }
                all_frame_done = true;
            }
        }

        __syncthreads();
        packet_reached = false;
        all_frame_done = false;
        if (threadIdx.x == 0) {
            sem_fr_recvinfo_idx = (sem_fr_recvinfo_idx + rx_pkt_num) % sem_fr_num;
        }
    }
}

template <typename T>
__inline__ __device__ T warpMax(T localMax)
{
    localMax = max(localMax, __shfl_xor_sync(0xffffffff, localMax, 16));
    localMax = max(localMax, __shfl_xor_sync(0xffffffff, localMax, 8));
    localMax = max(localMax, __shfl_xor_sync(0xffffffff, localMax, 4));
    localMax = max(localMax, __shfl_xor_sync(0xffffffff, localMax, 2));
    localMax = max(localMax, __shfl_xor_sync(0xffffffff, localMax, 1));

    return localMax;
}

#define MAX_THREAD_NUM (1024)
#define MAX_WARP_NUM (MAX_THREAD_NUM / 32)

__inline__ __device__ struct udp_payload_header get_seqn(uint8_t* p)
{
    struct udp_payload_header ret;
    memcpy(&(ret.seqn), p, sizeof(udp_payload_header));
    return ret;
}

__global__ void cuda_kernel_makeframe(
    uint8_t* tar_buf, size_t frame_size,
    uint8_t* tmp_buf,
    struct doca_gpu_eth_rxq* rxq,
    int sem_num, struct doca_gpu_semaphore_gpu* sem_recvinfo,
    uint64_t frame_num, struct doca_gpu_semaphore_gpu* sem_frame,
    bool is_warmup, int id)
{
    if (is_warmup) {
        if (threadIdx.x == 0) {
            printf("warmup cuda_kernel_makeframe\n");
        }
        return;
    }
    if (threadIdx.x == 0) {
        printf("cuda_kernel_makeframe performance\n");
    }

    __shared__ int64_t rx_buf_idx_head;
    __shared__ int64_t rx_buf_idx_tail;

    __shared__ bool packet_reached;
    __shared__ uint16_t packet_reached_thidx_share[MAX_WARP_NUM];
    uint16_t packet_reached_thidx = 0;

    __shared__ uint64_t frame_head;
    __shared__ uint32_t prev_ackn;
    __shared__ uint32_t next_prev_ackn;

    __shared__ uint8_t* cur_tar_buf;

    doca_error_t ret;
    struct doca_gpu_buf* buf_ptr;
    struct rx_info* rx_info_global;
    struct fr_info* fr_global;
    struct eth_ip_udp_hdr* hdr;
    uintptr_t buf_addr;
    __shared__ uint32_t sem_recvinfo_idx;
    __shared__ uint32_t sem_frame_idx;
    uint8_t* payload;
    __shared__ bool quit;
    int lane_id = threadIdx.x % warpSize;
    int warp_id = threadIdx.x / warpSize;

    frame_head = 0;
    if (threadIdx.x == 0) {
        prev_ackn = 0; // shared with client
        next_prev_ackn = 0;
        packet_reached = false;
        cur_tar_buf = nullptr;
        quit = false;
        sem_frame_idx = 0;
        sem_recvinfo_idx = 0;
    }

    if (blockIdx.x != 0) {
        return;
    }

    __syncthreads();

    enum doca_gpu_semaphore_status status;
    __shared__ enum doca_gpu_semaphore_status status_frame;

    // size_t heart_beat = 0;

    // size_t is_first = 0;

    // auto cl_start = clock();
    size_t bytes_local = 0;
    while ((!quit)) {

        // if (threadIdx.x == 0 && (heart_beat % ((size_t)100) == 0)) {
        //     printf("heartbeat frame %d\n", threadIdx.x);
        // }
        // heart_beat++;

        // auto cl_start = clock();
        while (true) {

            ret = doca_gpu_dev_semaphore_get_status(sem_recvinfo, (sem_recvinfo_idx + threadIdx.x) % sem_num, &status);
            if (ret != DOCA_SUCCESS) {
                printf("TCP semaphore error");
                return;
            }

            if (status == DOCA_GPU_SEMAPHORE_STATUS_READY) {
                packet_reached_thidx = threadIdx.x + 1;
            } else {
                packet_reached_thidx = 0;
            }

            __syncthreads();

            uint16_t local_max = warpMax(packet_reached_thidx);

            if (lane_id == 0) {
                packet_reached_thidx_share[warp_id] = local_max;
            }
            __syncthreads();

            if (threadIdx.x < warpSize) {
                local_max = threadIdx.x < MAX_WARP_NUM ? packet_reached_thidx_share[threadIdx.x] : 0;
                packet_reached_thidx_share[0] = warpMax(local_max);
            }

            __syncthreads();

            if (packet_reached_thidx_share[0] > 0) {

                if (warp_id == 0 && lane_id == 0) {

                    // printf("%d hoonto\n", packet_reached_thidx_share[0]);

                    ret = doca_gpu_dev_semaphore_get_custom_info_addr(sem_recvinfo,
                        sem_recvinfo_idx, (void**)&(rx_info_global));
                    if (ret != DOCA_SUCCESS) {
                        printf("TCP semaphore get address error\n");
                        return;
                    }

                    DOCA_GPUNETIO_VOLATILE(rx_buf_idx_head) = DOCA_GPUNETIO_VOLATILE(rx_info_global->rx_buf_idx);

                    __threadfence();

                    // printf("%d rx_pkt_num frame \n", rx_pkt_num);
                    // printf("%d rx_buf_idx frame \n", rx_buf_idx);

                    ret = doca_gpu_dev_semaphore_set_status(sem_recvinfo, sem_recvinfo_idx, DOCA_GPU_SEMAPHORE_STATUS_FREE);
                    if (ret != DOCA_SUCCESS) {
                        printf("TCP semaphore error\n");
                        return;
                    }
                    __threadfence_system();
                } else if (warp_id == 1 && lane_id == 0) {

                    size_t sem_idx = (sem_recvinfo_idx + packet_reached_thidx_share[0] - 1) % sem_num;

                    ret = doca_gpu_dev_semaphore_get_custom_info_addr(sem_recvinfo,
                        sem_idx, (void**)&(rx_info_global));
                    if (ret != DOCA_SUCCESS) {
                        printf("TCP semaphore get address error\n");
                        return;
                    }

                    int64_t rx_buf_idx;
                    int32_t rx_pkt_num;

                    DOCA_GPUNETIO_VOLATILE(rx_pkt_num) = DOCA_GPUNETIO_VOLATILE(rx_info_global->rx_pkt_num);
                    DOCA_GPUNETIO_VOLATILE(rx_buf_idx) = DOCA_GPUNETIO_VOLATILE(rx_info_global->rx_buf_idx);

                    __threadfence();
                    rx_buf_idx_tail = rx_buf_idx + rx_pkt_num;

                    // printf("%d rx_pkt_num frame \n", rx_pkt_num);
                    // printf("%d rx_buf_idx frame \n", rx_buf_idx);

                    ret = doca_gpu_dev_semaphore_set_status(sem_recvinfo, sem_idx, DOCA_GPU_SEMAPHORE_STATUS_FREE);
                    if (ret != DOCA_SUCCESS) {
                        printf("TCP semaphore error\n");
                        return;
                    }
                    __threadfence_system();
                } else if (warp_id >= 2) {
                    for (size_t i = threadIdx.x + 1 - 2 * warpSize; i < packet_reached_thidx_share[0] - 1; i += (blockDim.x - 2 * warpSize)) {
                        ret = doca_gpu_dev_semaphore_set_status(sem_recvinfo, (sem_recvinfo_idx + i) % sem_num, DOCA_GPU_SEMAPHORE_STATUS_FREE);
                        if (ret != DOCA_SUCCESS) {
                            printf("TCP semaphore error\n");
                            return;
                        }
                        __threadfence_system();
                    }
                }
                break;
            }
        }

        bool is_head_copy = false;

        if ((!cur_tar_buf) && threadIdx.x == 0) {
            ret = doca_gpu_dev_semaphore_get_status(sem_frame, sem_frame_idx, &status_frame);
            if (ret != DOCA_SUCCESS) {
                printf("TCP semaphore error");
                return;
            }
            if (status_frame == DOCA_GPU_SEMAPHORE_STATUS_FREE) {
                printf("%d %lld set buf\n", sem_frame_idx, frame_head);
                cur_tar_buf = tar_buf + sem_frame_idx * frame_size;
                is_head_copy = true;
            }
        }

        __syncthreads();

        if (threadIdx.x == 0) {
            sem_recvinfo_idx = (sem_recvinfo_idx + packet_reached_thidx_share[0]) % sem_num;
            if (is_head_copy)
                hipMemcpyAsync(cur_tar_buf, tmp_buf, frame_head, hipMemcpyDeviceToDevice);
            // if (heart_beat % 50 == 0) {
            //     printf("%d rx_buf_idx_head\n", rx_buf_idx_head);
            //     printf("%d rx_buf_idx_tail\n", rx_buf_idx_tail);
            // }
            // if (rx_buf_idx_tail >= MAX_PKT_NUM) {
            //     printf("-----------------------\n");
            //     printf("%d rx_buf_idx_head\n", rx_buf_idx_head);
            //     printf("%d rx_buf_idx_tail\n", rx_buf_idx_tail);
            // }
        }

        int64_t pkt_num = rx_buf_idx_tail >= rx_buf_idx_head ? rx_buf_idx_tail - rx_buf_idx_head : rx_buf_idx_tail - rx_buf_idx_head + MAX_PKT_NUM;
        int64_t pkt_remain = pkt_num % blockDim.x;
        int64_t pkt_num_local = pkt_num / blockDim.x;
        int64_t rx_buf_st = pkt_num_local * threadIdx.x;
        if (threadIdx.x < pkt_remain) {
            rx_buf_st += threadIdx.x;
            pkt_num_local++;
        } else {
            rx_buf_st += pkt_remain;
        }
        rx_buf_st += rx_buf_idx_head;

        // if (threadIdx.x == 0) {
        //     printf("%" PRIu64 " rx_buf_idx_head\n", rx_buf_idx_head);
        //     printf("%" PRIu64 " rx_buf_idx_tail\n", rx_buf_idx_tail);
        // }

        // if (is_first == 4) {
        //     printf("%" PRIu64 " rx_buf_st\n", rx_buf_st);
        // }
        // is_first++;

        if (cur_tar_buf) {
            bool is_printed = false;
            for (int64_t idx = rx_buf_st; idx < rx_buf_st + pkt_num_local; ++idx) {

                ret = doca_gpu_dev_eth_rxq_get_buf(rxq, idx % (int64_t)MAX_PKT_NUM, &buf_ptr);
                if (ret != DOCA_SUCCESS) {
                    printf("TCP Error %d doca_gpu_dev_eth_rxq_get_buf block %d thread %d\n", ret, blockIdx.x, threadIdx.x);
                    // DOCA_GPUNETIO_VOLATILE(*exit_cond) = 1;
                    break;
                }
                ret = doca_gpu_dev_buf_get_addr(buf_ptr, &buf_addr);
                if (ret != DOCA_SUCCESS) {
                    printf("TCP Error %d doca_gpu_dev_eth_rxq_get_buf block %d thread %d\n", ret, blockIdx.x, threadIdx.x);
                    // DOCA_GPUNETIO_VOLATILE(*exit_cond) = 1;
                    break;
                }
                raw_to_udp(buf_addr, &hdr, &payload);
                uint32_t sent_seq = get_seqn(payload).seqn;
                payload += sizeof(struct udp_payload_header);
                uint32_t total_payload_size = BYTE_SWAP16(hdr->l4_hdr.dgram_len) - sizeof(struct udp_hdr) - sizeof(struct udp_payload_header);

                if (idx == rx_buf_idx_tail - 1) {
                    next_prev_ackn = sent_seq + total_payload_size;
                }

                uint32_t offset = sent_seq - prev_ackn;
                uint64_t cur_head = frame_head + offset;

                if (cur_head + total_payload_size <= frame_size) {
                    uint32_t write_byte = total_payload_size;
                    uint8_t* data_head = cur_tar_buf + cur_head;
                    hipMemcpyAsync(data_head, payload, write_byte, hipMemcpyDeviceToDevice);
                } else if (cur_head < frame_size) {
                    uint32_t write_byte = frame_size - cur_head;
                    uint8_t* data_head = cur_tar_buf + cur_head;
                    hipMemcpyAsync(data_head, payload, write_byte, hipMemcpyDeviceToDevice);
                    hipMemcpyAsync(tmp_buf, payload + write_byte, total_payload_size - write_byte, hipMemcpyDeviceToDevice);
                    // if (total_payload_size - write_byte > (size_t)1 * (size_t)1024 * 1024 * 1024) {
                    //     printf("kokokoko\n");
                    // }
                } else {
                    hipMemcpyAsync(tmp_buf + cur_head - frame_size, payload, total_payload_size, hipMemcpyDeviceToDevice);
                    if ((!is_printed) && cur_head - frame_size + total_payload_size > (size_t)1 * (size_t)512 * 1024 * 1024) {
                        printf("%" PRIx64 " idx\n", idx);
                        printf("%" PRIu64 " idx_round\n", idx % MAX_PKT_NUM);
                        printf("%" PRIu64 " sent_seq\n", sent_seq);
                        printf("%" PRIu64 " prev_ackn\n", prev_ackn);
                        is_printed = true;
                    }
                }
            }
        }

        // for (int th_num = (blockDim.x + warpSize - 1) / warpSize; th_num > 1; th_num = (th_num + warpSize - 1) / warpSize) {
        //     if (threadIdx.x < warpSize * ((th_num + warpSize - 1) / warpSize)) {
        //         local_max = threadIdx.x < th_num ? packet_reached_thidx_share[threadIdx.x] : 0;
        //         packet_reached_thidx_share[threadIdx.x / warpSize] = warpMax(local_max);
        //     }
        //     __syncthreads();
        // }
        __syncthreads();

        if (warp_id == 1 && lane_id == 0) {
            uint64_t bytes = (next_prev_ackn - prev_ackn);
            // printf("%" PRIu64 " prev_ackn\n", bytes);
            // bytes_local += bytes;
            // if (heart_beat % 50 == 0) {
            //     auto cl_end = clock();
            //     // printf("%" PRIu64 " bytes\n", bytes_local);
            //     printf("%lf %d Gbps\n", 8 * bytes_local / ((cl_end - cl_start) / (1.5)), (cl_end - cl_start));
            //     cl_start = clock();
            //     bytes_local = 0;
            // }
            frame_head += bytes;
            if (frame_head > 2 * frame_size) {
                printf("error\n");
            }
            if (frame_head > frame_size) {
                ret = doca_gpu_dev_semaphore_get_custom_info_addr(sem_frame, sem_frame_idx, (void**)&(fr_global));
                DOCA_GPUNETIO_VOLATILE(fr_global->eth_payload) = DOCA_GPUNETIO_VOLATILE(cur_tar_buf);
                __threadfence_system();

                ret = doca_gpu_dev_semaphore_set_status(sem_frame, sem_frame_idx, DOCA_GPU_SEMAPHORE_STATUS_READY);
                __threadfence_system();
                printf("%llu %u frame_head send\n", frame_head, packet_reached_thidx_share[0]);
                printf("%u %d pkt_num\n", pkt_num, id);
                sem_frame_idx = (sem_frame_idx + 1) % frame_num;
                cur_tar_buf = nullptr;
                frame_head -= frame_size;
                // quit = true;
            }
            prev_ackn = next_prev_ackn;
        }

        __syncthreads();
        packet_reached = false;
    }
}

__inline__ __device__ void swap_eth(struct ether_hdr* eth)
{
    uint8_t tmp_addr[ETHER_ADDR_LEN];
    memcpy(tmp_addr, eth->d_addr_bytes, ETHER_ADDR_LEN);
    memcpy(eth->d_addr_bytes, eth->s_addr_bytes, ETHER_ADDR_LEN);
    memcpy(eth->s_addr_bytes, tmp_addr, ETHER_ADDR_LEN);
}

__global__ void cuda_kernel_send_packets(
    struct doca_gpu_eth_txq* txq,
    struct doca_gpu_buf_arr* tx_buf_arr,
    int sem_reply_num, struct doca_gpu_semaphore_gpu* sem_reply_recvinfo,
    bool is_warmup)
{
    if (is_warmup) {
        if (threadIdx.x == 0) {
            printf("warmup cuda_kernel_send_packets\n");
        }
        return;
    }

    doca_error_t ret;
    struct doca_gpu_buf* buf_ptr;
    struct reply_info* reply_info_global;
    struct eth_ip_udp_hdr* hdr;
    uint8_t* buf_addr;
    __shared__ uint32_t sem_reply_recvinfo_idx;
    enum doca_gpu_semaphore_status reply_status;
    __shared__ int32_t max_sent_id[32];
    const uint32_t base_pkt_len = sizeof(struct eth_ip_udp_hdr);

    int warp_id = threadIdx.x / warpSize;
    int lane_id = threadIdx.x % warpSize;
    int warp_num = blockDim.x / warpSize;
    int th_id = threadIdx.x;

    if (threadIdx.x == 0) {
        sem_reply_recvinfo_idx = 0;
    }

    __syncthreads();

    while (true) {
        uint32_t sem_idx = (sem_reply_recvinfo_idx + threadIdx.x) % sem_reply_num;
        ret = doca_gpu_dev_semaphore_get_status(sem_reply_recvinfo, sem_idx, &reply_status);
        if (ret != DOCA_SUCCESS) {
            printf("TCP semaphore error");
            return;
        }
        int32_t sent_id = -1;
        if (reply_status == DOCA_GPU_SEMAPHORE_STATUS_READY) {
            sent_id = threadIdx.x;
            ret = doca_gpu_dev_semaphore_get_custom_info_addr(sem_reply_recvinfo, sem_idx, (void**)&(reply_info_global));
            if (ret != DOCA_SUCCESS) {
                printf("TCP semaphore get address error\n");
                return;
            }

            DOCA_GPUNETIO_VOLATILE(buf_addr) = DOCA_GPUNETIO_VOLATILE(reply_info_global->eth_payload);

            __threadfence();

            // printf("%d rx_pkt_num frame \n", rx_pkt_num);
            // printf("%d rx_buf_idx frame \n", rx_buf_idx);

            ret = doca_gpu_dev_semaphore_set_status(sem_reply_recvinfo, sem_idx, DOCA_GPU_SEMAPHORE_STATUS_FREE);
            if (ret != DOCA_SUCCESS) {
                printf("TCP semaphore error\n");
                return;
            }
            __threadfence_system();

            struct doca_gpu_buf* reply_buf = NULL;
            ret = doca_gpu_dev_buf_get_buf(tx_buf_arr, threadIdx.x, &reply_buf);

            uintptr_t reply_buf_addr;
            ret = doca_gpu_dev_buf_get_addr(reply_buf, &reply_buf_addr);

            hdr = (struct eth_ip_udp_hdr*)buf_addr;

            memcpy((uint8_t*)reply_buf_addr, buf_addr, BYTE_SWAP16(hdr->l3_hdr.total_length) + sizeof(struct ether_hdr));

            hdr = (struct eth_ip_udp_hdr*)reply_buf_addr;

            swap_eth(&(hdr->l2_hdr));

            auto tmp_src_addr = hdr->l3_hdr.src_addr;
            hdr->l3_hdr.src_addr = hdr->l3_hdr.dst_addr;
            hdr->l3_hdr.dst_addr = tmp_src_addr;
            auto tmp_src_port = hdr->l4_hdr.src_port;
            hdr->l4_hdr.src_port = hdr->l4_hdr.dst_port;
            hdr->l4_hdr.dst_port = tmp_src_port;

            // printf("%d l4_hdr bytes send", BYTE_SWAP16(hdr->l4_hdr.dgram_len));

            ret = doca_gpu_dev_eth_txq_send_enqueue_strong(txq, reply_buf, base_pkt_len + BYTE_SWAP16(hdr->l4_hdr.dgram_len) - sizeof(udp_hdr), 0);
            if (ret != DOCA_SUCCESS) {
                printf("Error %d doca_gpu_dev_eth_txq_send_enqueue_strong block %d thread %d\n", ret, warp_id, lane_id);
                // DOCA_GPUNETIO_VOLATILE(*exit_cond) = 1;
                break;
            }
        }

        sent_id = warpMax(sent_id);
        if (lane_id == 0) {
            max_sent_id[warp_id] = sent_id;
        }
        __syncthreads();
        if (th_id < warpSize) {
            sent_id = th_id < warp_num ? max_sent_id[th_id] : -1;
        }
        if (warp_id == 0) {
            sent_id = warpMax(sent_id);
        }

        if (threadIdx.x == 0 && sent_id >= 0) {
            doca_gpu_dev_eth_txq_commit_strong(txq);
            doca_gpu_dev_eth_txq_push(txq);
        }

        if (threadIdx.x == 0 && sent_id > 0) {
            sem_reply_recvinfo_idx += sent_id;
        }

        __syncthreads();
    }
}

void init_udp_echo_kernels(std::vector<hipStream_t>& streams)
{
    cuda_kernel_receive_udp<<<1, CUDA_THREADS>>>(
        nullptr, 0, nullptr, true);
    cuda_kernel_makeframe_echo_server<<<1, CUDA_THREADS>>>(
        nullptr, 0, nullptr, 0, nullptr, true);
    cuda_kernel_send_packets<<<1, CUDA_THREADS>>>(
        nullptr, nullptr, 0, nullptr, true);

    streams.resize(3);

    hipStreamCreate(&streams[0]);
    hipStreamCreate(&streams[1]);
    hipStreamCreate(&streams[2]);
}

void launch_udp_echo_kernels(struct rx_queue* rxq,
    struct tx_queue* txq,
    struct tx_buf* tx_buf_arr,
    struct semaphore* sem_rx,
    struct semaphore* sem_fr,
    struct semaphore* sem_reply,
    std::vector<hipStream_t>& streams)
{
    cuda_kernel_receive_udp<<<1, CUDA_THREADS, 0, streams.at(0)>>>(
        rxq->eth_rxq_gpu,
        sem_rx->sem_num,
        sem_rx->sem_gpu, false);

    cuda_kernel_makeframe_echo_server<<<1, CUDA_THREADS, 0, streams.at(1)>>>(
        rxq->eth_rxq_gpu,
        sem_rx->sem_num, sem_rx->sem_gpu,
        sem_fr->sem_num, sem_fr->sem_gpu,
        false);

    cuda_kernel_send_packets<<<1, CUDA_THREADS, 0, streams.at(2)>>>(
        txq->eth_txq_gpu, tx_buf_arr->buf_arr_gpu, sem_reply->sem_num, sem_reply->sem_gpu,
        false);
}

void init_udp_framebuilding_kernels(std::vector<hipStream_t>& streams)
{
    cuda_kernel_receive_udp<<<1, CUDA_THREADS>>>(
        nullptr, 0, nullptr, true);

    cuda_kernel_makeframe<<<1, CUDA_THREADS>>>(
        nullptr, 0, nullptr,
        nullptr, 0, nullptr, 0, nullptr, true, 0);

    streams.resize(2);

    hipStreamCreate(&streams[0]);
    hipStreamCreate(&streams[1]);
}

void launch_udp_framebuilding_kernels(struct rx_queue* rxq,
    struct semaphore* sem_rx,
    struct semaphore* sem_fr,
    uint8_t* tar_buf, size_t frame_size,
    uint8_t* tmp_buf,
    std::vector<hipStream_t>& streams)
{
    cuda_kernel_receive_udp<<<1, 32, 0, streams.at(0)>>>(
        rxq->eth_rxq_gpu,
        sem_rx->sem_num,
        sem_rx->sem_gpu, false);

    cuda_kernel_makeframe<<<1, MAX_THREAD_NUM, 0, streams.at(1)>>>(
        tar_buf, frame_size,
        tmp_buf,
        rxq->eth_rxq_gpu,
        sem_rx->sem_num, sem_rx->sem_gpu,
        sem_fr->sem_num, sem_fr->sem_gpu,
        false, 0);
}

}
